#include <iostream>
#include <hip/hip_runtime.h>

void getPcieInfo(int deviceId) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);

    int pciBusId, pciDeviceId, pciDomainId;
    hipDeviceGetAttribute(&pciBusId, hipDeviceAttributePciBusId, deviceId);
    hipDeviceGetAttribute(&pciDeviceId, hipDeviceAttributePciDeviceId, deviceId);
    hipDeviceGetAttribute(&pciDomainId, hipDeviceAttributePciDomainId, deviceId);

    int pciLinkGeneration;
    hipDeviceGetAttribute(&pciLinkGeneration, hipDeviceAttributePciBusId, deviceId);

    int pciLinkWidth;
    hipDeviceGetAttribute(&pciLinkWidth, hipDeviceAttributePciBusId, deviceId);

    std::cout << "Device " << deviceId << ": " << deviceProp.name << "\n";
    std::cout << "  PCI Bus ID: " << pciBusId << "\n";
    std::cout << "  PCI Device ID: " << pciDeviceId << "\n";
    std::cout << "  PCI Domain ID: " << pciDomainId << "\n";
    std::cout << "  PCIe Link Generation: " << pciLinkGeneration << "\n";
    std::cout << "  PCIe Link Width: " << pciLinkWidth << "\n";
}

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found.\n";
        return 1;
    }

    for (int deviceId = 0; deviceId < deviceCount; ++deviceId) {
        getPcieInfo(deviceId);
    }

    return 0;
}
